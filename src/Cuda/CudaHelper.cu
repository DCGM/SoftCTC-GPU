#include "CudaHelper.h"

//Vrati retezec pro opencl error kod
const char *get_cuda_error_string(hipError_t msg_id)
{
    return(hipGetErrorString(msg_id));
  /*switch (msg_id)
  {
  case hipSuccess:
    return "hipSuccess";
  case hipErrorInvalidValue:
    return "hipErrorInvalidValue";
  case hipErrorOutOfMemory:
    return "hipErrorOutOfMemory";
  case hipErrorNotInitialized:
    return "hipErrorNotInitialized";
  case hipErrorDeinitialized:
    return "hipErrorDeinitialized";
  case hipErrorProfilerDisabled:
    return "hipErrorProfilerDisabled";
  case hipErrorProfilerNotInitialized:
    return "hipErrorProfilerNotInitialized";
  case hipErrorProfilerAlreadyStarted:
    return "hipErrorProfilerAlreadyStarted";
  case hipErrorProfilerAlreadyStopped:
    return "hipErrorProfilerAlreadyStopped";
  case hipErrorInvalidConfiguration:
    return "hipErrorInvalidConfiguration";
  case hipErrorInvalidPitchValue:
    return "hipErrorInvalidPitchValue";
  case hipErrorInvalidSymbol:
    return "hipErrorInvalidSymbol";
  case cudaErrorInvalidHostPointer:
    return "cudaErrorInvalidHostPointer";
  case hipErrorInvalidDevicePointer:
    return "hipErrorInvalidDevicePointer";
  case hipErrorInvalidTexture:
    return "hipErrorInvalidTexture";
  case cudaErrorInvalidTextureBinding:
    return "cudaErrorInvalidTextureBinding";
  case hipErrorInvalidChannelDescriptor:
    return "hipErrorInvalidChannelDescriptor";
  case hipErrorInvalidMemcpyDirection:
    return "hipErrorInvalidMemcpyDirection";
  case cudaErrorAddressOfConstant:
    return "cudaErrorAddressOfConstant";
  case cudaErrorTextureFetchFailed:
    return "cudaErrorTextureFetchFailed";
  case cudaErrorTextureNotBound:
    return "cudaErrorTextureNotBound";
  case cudaErrorSynchronizationError:
    return "cudaErrorSynchronizationError";
  case cudaErrorInvalidFilterSetting:
    return "cudaErrorInvalidFilterSetting";
  case cudaErrorInvalidNormSetting:
    return "cudaErrorInvalidNormSetting";
  case cudaErrorMixedDeviceExecution:
    return "cudaErrorMixedDeviceExecution";
  case cudaErrorNotYetImplemented:
    return "cudaErrorNotYetImplemented";
  case cudaErrorMemoryValueTooLarge:
    return "cudaErrorMemoryValueTooLarge";
  case cudaErrorStubLibrary:
    return "cudaErrorStubLibrary";
  case hipErrorInsufficientDriver:
    return "hipErrorInsufficientDriver";
  case cudaErrorCallRequiresNewerDriver:
    return "cudaErrorCallRequiresNewerDriver";
  case cudaErrorInvalidSurface:
    return "cudaErrorInvalidSurface";
  case cudaErrorDuplicateVariableName:
    return "cudaErrorDuplicateVariableName";
  case cudaErrorDuplicateTextureName:
    return "cudaErrorDuplicateTextureName";
  case cudaErrorDuplicateSurfaceName:
    return "cudaErrorDuplicateSurfaceName";
  case cudaErrorDevicesUnavailable:
    return "cudaErrorDevicesUnavailable";
  case cudaErrorIncompatibleDriverContext:
    return "cudaErrorIncompatibleDriverContext";
  case hipErrorMissingConfiguration:
    return "hipErrorMissingConfiguration";
  case hipErrorPriorLaunchFailure:
    return "hipErrorPriorLaunchFailure";
  case cudaErrorLaunchMaxDepthExceeded:
    return "cudaErrorLaunchMaxDepthExceeded";
  case cudaErrorLaunchFileScopedTex:
    return "cudaErrorLaunchFileScopedTex";
  case cudaErrorLaunchFileScopedSurf:
    return "cudaErrorLaunchFileScopedSurf";
  case cudaErrorSyncDepthExceeded:
    return "cudaErrorSyncDepthExceeded";
  case cudaErrorLaunchPendingCountExceeded:
    return "cudaErrorLaunchPendingCountExceeded";
  case hipErrorInvalidDeviceFunction:
    return "hipErrorInvalidDeviceFunction";
  case hipErrorNoDevice:
    return "hipErrorNoDevice";
  case hipErrorInvalidDevice:
    return "hipErrorInvalidDevice";
  case cudaErrorDeviceNotLicensed:
      return "cudaErrorDeviceNotLicensed";
  case cudaErrorSoftwareValidityNotEstablished:
      return "cudaErrorSoftwareValidityNotEstablished";
  case cudaErrorStartupFailure:
      return "cudaErrorStartupFailure";
  case hipErrorInvalidImage:
      return "hipErrorInvalidImage";
  case hipErrorInvalidContext:
      return "hipErrorInvalidContext";
  case hipErrorMapFailed:
      return "hipErrorMapFailed";
  case hipErrorUnmapFailed:
      return "hipErrorUnmapFailed";
  case hipErrorArrayIsMapped:
      return "hipErrorArrayIsMapped";
  case hipErrorAlreadyMapped:
      return "hipErrorAlreadyMapped";
  case hipErrorNoBinaryForGpu:
      return "hipErrorNoBinaryForGpu";
  case hipErrorAlreadyAcquired:
      return "hipErrorAlreadyAcquired";
  case hipErrorNotMapped:
      return "hipErrorNotMapped";
  case hipErrorNotMappedAsArray:
      return "hipErrorNotMappedAsArray";
  case hipErrorNotMappedAsPointer:
      return "hipErrorNotMappedAsPointer";
  case hipErrorECCNotCorrectable:
      return "hipErrorECCNotCorrectable";
  case hipErrorUnsupportedLimit:
      return "hipErrorUnsupportedLimit";
  case hipErrorContextAlreadyInUse:
      return "hipErrorContextAlreadyInUse";
  case hipErrorPeerAccessUnsupported:
      return "hipErrorPeerAccessUnsupported";
  case hipErrorInvalidKernelFile:
      return "hipErrorInvalidKernelFile";
  case hipErrorInvalidGraphicsContext:
      return "hipErrorInvalidGraphicsContext";
  case cudaErrorNvlinkUncorrectable:
      return "cudaErrorNvlinkUncorrectable";
  case cudaErrorJitCompilerNotFound:
      return "cudaErrorJitCompilerNotFound";
  case cudaErrorUnsupportedPtxVersion:
      return "cudaErrorUnsupportedPtxVersion";
  case cudaErrorJitCompilationDisabled:
      return "cudaErrorJitCompilationDisabled";
  case hipErrorInvalidSource:
      return "hipErrorInvalidSource";
  case hipErrorFileNotFound:
      return "hipErrorFileNotFound";
  case hipErrorSharedObjectSymbolNotFound:
      return "hipErrorSharedObjectSymbolNotFound";
  case hipErrorSharedObjectInitFailed:
      return "hipErrorSharedObjectInitFailed";
  case hipErrorOperatingSystem:
      return "hipErrorOperatingSystem";
  case hipErrorInvalidHandle:
      return "hipErrorInvalidHandle";
  case hipErrorIllegalState:
      return "hipErrorIllegalState";
  case hipErrorNotFound:
      return "hipErrorNotFound";
  case hipErrorNotReady:
      return "hipErrorNotReady";
  case hipErrorIllegalAddress:
      return "hipErrorIllegalAddress";
  case hipErrorLaunchOutOfResources:
      return "hipErrorLaunchOutOfResources";
  case hipErrorLaunchTimeOut:
      return "hipErrorLaunchTimeOut";
  case cudaErrorLaunchIncompatibleTexturing:
      return "cudaErrorLaunchIncompatibleTexturing";
  case hipErrorPeerAccessAlreadyEnabled:
      return "hipErrorPeerAccessAlreadyEnabled";
  case hipErrorPeerAccessNotEnabled:
      return "hipErrorPeerAccessNotEnabled";
  case hipErrorSetOnActiveProcess:
      return "hipErrorSetOnActiveProcess";
  case hipErrorContextIsDestroyed:
      return "hipErrorContextIsDestroyed";
  case hipErrorAssert:
      return "hipErrorAssert";
  case cudaErrorTooManyPeers:
      return "cudaErrorTooManyPeers";
  case hipErrorHostMemoryAlreadyRegistered:
      return "hipErrorHostMemoryAlreadyRegistered";
  case hipErrorHostMemoryNotRegistered:
      return "hipErrorHostMemoryNotRegistered";
  case cudaErrorHardwareStackError:
      return "cudaErrorHardwareStackError";
  case cudaErrorIllegalInstruction:
      return "cudaErrorIllegalInstruction";
  case cudaErrorMisalignedAddress:
      return "cudaErrorMisalignedAddress";
  case cudaErrorInvalidAddressSpace:
      return "cudaErrorInvalidAddressSpace";
  case cudaErrorInvalidPc:
      return "cudaErrorInvalidPc";
  case hipErrorLaunchFailure:
      return "hipErrorLaunchFailure";
  case hipErrorCooperativeLaunchTooLarge:
      return "hipErrorCooperativeLaunchTooLarge";
  case cudaErrorNotPermitted:
      return "cudaErrorNotPermitted";
  case hipErrorNotSupported:
      return "hipErrorNotSupported";
  case cudaErrorSystemNotReady:
      return "cudaErrorSystemNotReady";
  case cudaErrorSystemDriverMismatch:
      return "cudaErrorSystemDriverMismatch";
  case cudaErrorCompatNotSupportedOnDevice:
      return "cudaErrorCompatNotSupportedOnDevice";
  case hipErrorStreamCaptureUnsupported:
      return "hipErrorStreamCaptureUnsupported";
  case hipErrorStreamCaptureInvalidated:
      return "hipErrorStreamCaptureInvalidated";
  case hipErrorStreamCaptureMerge:
      return "hipErrorStreamCaptureMerge";
  case hipErrorStreamCaptureUnmatched:
      return "hipErrorStreamCaptureUnmatched";
  case hipErrorStreamCaptureUnjoined:
      return "hipErrorStreamCaptureUnjoined";
  case hipErrorStreamCaptureIsolation:
      return "hipErrorStreamCaptureIsolation";
  case hipErrorStreamCaptureImplicit:
      return "hipErrorStreamCaptureImplicit";
  case hipErrorCapturedEvent:
      return "hipErrorCapturedEvent";
  case hipErrorStreamCaptureWrongThread:
      return "hipErrorStreamCaptureWrongThread";
  case cudaErrorTimeout:
      return "cudaErrorTimeout";
  case hipErrorGraphExecUpdateFailure:
      return "hipErrorGraphExecUpdateFailure";
  case hipErrorUnknown:
      return "hipErrorUnknown";
  case cudaErrorApiFailureBase:
      return "cudaErrorApiFailureBase";
  default:
    return "Unknown";
  }*/
}

bool cudaPrintErrorExit(hipError_t err_num, const char *text)
{
  if(err_num != hipSuccess)
  {
    std::cerr << "Error: " << text << ": (" << err_num << ") " << get_cuda_error_string(err_num) << std::endl;
    std::string input_data;
    std::cin >> input_data;
    exit(1);
    return false;
  }
  return true;
}

bool cudaPrintError(hipError_t err_num, const char *text, std::ostream *error_stream)
{
  if((err_num != hipSuccess) && (error_stream != NULL)) *error_stream << "Error: " << text << ": (" << err_num << ") " << get_cuda_error_string(err_num) << std::endl;
  return (err_num == hipSuccess);
}

bool cudaPrintInfo(hipError_t err_num, const char *text, std::ostream *error_stream)
{
  if((err_num != hipSuccess) && (error_stream != NULL)) *error_stream << "Info: " << text << ": (" << err_num << ") " << get_cuda_error_string(err_num) << std::endl;
  return (err_num == hipSuccess);
}

double getCudaEventTime(hipEvent_t start, hipEvent_t stop)
{
    float ms;
    hipEventElapsedTime(&ms, start, stop);
    return ms/1000.0;
}